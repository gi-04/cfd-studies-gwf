
#include <hip/hip_runtime.h>
#include<stdio.h>

__global__
void print_sizes();

int main(){

	print_sizes<<<2,2>>>();
	
	return 0;
}

__global__
void print_sizes(){
	printf("block indexes: %d %d %d\n",blockIdx.x,blockIdx.y,blockIdx.z);
	printf("thread indexes: %d %d %d\n",threadIdx.x,threadIdx.y,threadIdx.z);
	printf("grid dimensions: %d, %d, %d\n",gridDim.x,gridDim.y,gridDim.z);
	printf("block dimensions: %d, %d, %d\n",blockDim.x,blockDim.y,blockDim.z);
}