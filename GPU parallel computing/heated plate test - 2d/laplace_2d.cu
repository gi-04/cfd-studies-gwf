
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<iostream>

/*
a alterar:
-solucionador não está sendo eficiente-creio que o mesmo processo é repetido em
 múltiplas threads, o que não é necessário
-resolver a questão dos resíduos
*/


void print_2d_array(int Nx,int Ny,int k,double *M){
  for(int j=Ny-1;j>=0;j--){
    for(int i=0;i<Nx;i++)
      printf("%f ",M[k*Nx*Ny+Ny*j+i]);

    putchar('\n');
  }
}

__device__
void dev_print_2d_array(int Nx,int Ny,int k,double *M){
  for(int j=Ny-1;j>=0;j--){
    for(int i=0;i<Nx;i++)
      printf("%f ",M[k*Nx*Ny+Ny*j+i]);

    printf("\n");
  }
}

// void print_2d_array(int Nx,int Ny,int k,double *M){
  // for(int j=0;j<Ny;j++){
    // for(int i=0;i<Nx;i++)
      // printf("%f ",M[k*Nx*Ny+Ny*j+i]);

    // putchar('\n');
  // }
// }

__global__
void initialize(int size,double *M){
	int index = blockIdx.x*blockDim.x + threadIdx.x;
  int stride = blockDim.x*gridDim.x;
  for(int i=index;i<size;i+=stride){
    M[i] = 0;
		// printf("%f ",M[i]);
	}
}

__device__
void solver(int Nx,int Ny,double *sol,int k,double beta){
	for(int j=1;j<Ny-1;j++){
		for(int i=1;i<Nx-1;i++){
			sol[k*Nx*Ny+Ny*j+i] = 1./(2.*(1.+beta*beta))*(sol[k*Nx*Ny+Ny*j+(i+1)] + 
														sol[k*Nx*Ny+Ny*j+(i-1)] + 
														beta*beta*sol[k*Nx*Ny+Ny*(j+1)+i] + 
														beta*beta*sol[k*Nx*Ny+Ny*(j-1)+i]);
		}
	}
}

// Regions are considered as below:
// | [0] [1] |
// | [2] [3] | 
__global__
void physical_boundary_conditions(int Nxs,int Nys,double *sol,double T_left,
																	double T_right,double T_up,double T_down){
	
	int index = blockIdx.x*blockDim.x + threadIdx.x;
  int stride = blockDim.x*gridDim.x;
	for(int j=index;j<Nys-1;j+=stride){
		sol[0*Nxs*Nys+Nys*j+0] = T_left;
		sol[2*Nxs*Nys+Nys*j+0] = T_left;
		sol[1*Nxs*Nys+Nys*j+(Nxs-1)] = T_right;
		sol[3*Nxs*Nys+Nys*j+(Nxs-1)] = T_right;
  }
  for(int i=index;i<Nxs-1;i+=stride){
		sol[2*Nxs*Nys+Nys*0+i] = T_down;
		sol[3*Nxs*Nys+Nys*0+i] = T_down;
		sol[0*Nxs*Nys+Nys*(Nys-1)+i] = T_up;
		sol[1*Nxs*Nys+Nys*(Nys-1)+i] = T_up;
  }
}

__global__
void phantom_boundary_conditions(int Nxs,int Nys,double *sol){
	int index = blockIdx.x*blockDim.x + threadIdx.x;
  int stride = blockDim.x*gridDim.x;
	// Vertical
	for(int j=index;j>0&&j<Nys;j+=stride){
		// Left to right
		sol[3*Nxs*Nys+Nys*j+0] = sol[2*Nxs*Nys+Nys*j+(Nxs-2)];
		sol[1*Nxs*Nys+Nys*(j-1)+0] = sol[0*Nxs*Nys+Nys*(j-1)+(Nxs-2)];
		
		// Right to left
		sol[2*Nxs*Nys+Nys*j+(Nxs-1)] = sol[3*Nxs*Nys+Nys*j+1];
		sol[0*Nxs*Nys+Nys*(j-1)+(Nxs-1)] = sol[1*Nxs*Nys+Nys*(j-1)+1];
	}
	
	// Horizontal
	for(int i=index;i>0&&i<Nxs;i+=stride){
		// Lower to upper
		sol[0*Nxs*Nys+Nys*0+i] = sol[2*Nxs*Nys+Nys*(Nys-2)+i];
		sol[1*Nxs*Nys+Nys*0+(i-1)] = sol[3*Nxs*Nys+Nys*(Nys-2)+(i-1)];
		
		// Upper to lower
		sol[2*Nxs*Nys+Nys*(Nys-1)+i] = sol[0*Nxs*Nys+Nys*1+i];
		sol[3*Nxs*Nys+Nys*(Nys-1)+(i-1)] = sol[1*Nxs*Nys+Nys*1+(i-1)];	
	}
}

__device__
void save_sol_old_loop2(int Nxs,int Nys,double *sol,double *sol_old,int k,
												int j){
	int index = blockIdx.x*blockDim.x + threadIdx.x;
  int stride = blockDim.x*gridDim.x;
	for(int i=index;i>0&&i<Nxs-1;i+=stride)
		sol_old[k*(Nxs-2)*(Nys-2)+(Nys-2)*(j-1)+(i-1)] = sol[k*Nxs*Nys+Nys*j+i];
}

__device__
void save_sol_old(int Nxs,int Nys,double *sol,double *sol_old,int k){
	int index = blockIdx.x*blockDim.x + threadIdx.x;
  int stride = blockDim.x*gridDim.x;
	for(int j=index;j>0&&j<Nys-1;j+=stride)
		save_sol_old_loop2(Nxs,Nys,sol,sol_old,k,j);
}

__device__
void get_residuals_loop2(int Nxs,int Nys,double *sol,double *sol_old,double *res,
									 int k,int j){
	int index = blockIdx.x*blockDim.x + threadIdx.x;
  int stride = blockDim.x*gridDim.x;
	for(int i=index;i>0&&i<Nxs-1;i+=stride)
		res[k*(Nxs-2)*(Nys-2)+(Nys-2)*(j-1)+(i-1)] = fabs(sol[k*Nxs*Nys+Nys*j+i] - 
							 sol_old[k*(Nxs-2)*(Nys-2)+(Nys-2)*(j-1)+(i-1)]);
}

__device__
void get_residuals(int Nxs,int Nys,double *sol,double *sol_old,double *res,
									 int k){
	int index = blockIdx.x*blockDim.x + threadIdx.x;
  int stride = blockDim.x*gridDim.x;
	for(int j=index;j>0&&j<Nys-1;j+=stride)
		get_residuals_loop2(Nxs,Nys,sol,sol_old,res,k,j);
}

double sum_residuals(int Nxs,int Nys,double *res){
	double res_sum = 0.;
	for(int i=0;i<Nxs*Nys*4;i++)
		res_sum += res[i];
	
	return res_sum;
}

__global__ 
void iterate_once(int Nxs,int Nys,double *sol,double *sol_old,double *res,
									double beta){
	int index = blockIdx.x*blockDim.x + threadIdx.x;
  int stride = blockDim.x*gridDim.x;
	// printf("iterate_once: %d %d\n",index,stride);
	for(int k=index;k<4;k+=stride){
		save_sol_old(Nxs,Nys,sol,sol_old,k); // Save current values for convergence checking
		// dev_print_2d_array(Nxs-2,Nys-2,k,sol_old);
		solver(Nxs,Nys,sol,k,beta);
		get_residuals(Nxs,Nys,sol,sol_old,res,k);
	}
}

// __global__
void parallel_solver(int Nxs,int Nys,double *sol,double *sol_old,double *res,
										 int iter,double beta,double eps,int blocksPerGrid,
										 int threadsPerBlock){
	double res_val;
	for(int loop=0;loop<iter;loop++){
		// Calculate one iteration for each region
		iterate_once<<<blocksPerGrid,threadsPerBlock>>>(Nxs,Nys,sol,sol_old,res,beta);
		hipDeviceSynchronize();
		
		for(int k=0;k<4;k++){
			printf("%d\n",k);
			print_2d_array(Nxs,Nys,k,sol);
			putchar('\n');
			print_2d_array(Nxs-2,Nys-2,k,sol_old);
			putchar('\n');
			print_2d_array(Nxs-2,Nys-2,k,res);
			putchar('\n');
		}
		getchar();
		
		// Check convergence
		res_val = sum_residuals(Nxs-2,Nys-2,res);
		if(res_val <= eps){
			puts("Convergence!");
			break;
		}
		
		printf("Iteration %d | Residuals = %f\n",loop+1,res_val);
		
		// Update phantom boundary conditions
		phantom_boundary_conditions<<<blocksPerGrid,threadsPerBlock>>>(Nxs,Nys,sol);
		hipDeviceSynchronize();
	}
	
	// cudaFree(res_val);
}

int main(){
	time_t start,end;
	time(&start);
	
	// Problem parameters
  double Lx = 1., Ly = 1.;
  double T_down = 0., T_up = 0.;
  double T_left = 50., T_right = 50.;

  // Mesh parameters
  int Nx = 10, Ny = 10; // Complete domain (even values)
  double deltaX = Lx/((double) Nx);
  double deltaY = Ly/((double) Ny);
  double beta = deltaX/deltaY;
	
	// Numeric solution configuration
  int iter = 50000;
  double eps = 1e-3;
	
	// Initialize arrays
	Nx = (int) (Nx/2)*2; // Ensure values are even
	Ny = (int) (Ny/2)*2;
	int Nxs = Nx/2+1; // Dimensions for the submatrices
	int Nys = Ny/2+1;
	double *sol = new double[4*Nxs*Nys];
	double *sol_old = new double[4*(Nxs-2)*(Nys-2)];
	double *res = new double[4*(Nxs-2)*(Nys-2)];
	hipMallocManaged(&sol,sizeof(double)*4*Nxs*Nys);
	hipMallocManaged(&sol_old,sizeof(double)*4*(Nxs-2)*(Nys-2));
	hipMallocManaged(&res,sizeof(double)*4*(Nxs-2)*(Nys-2));
	
	// Allocate relevant arrays in GPU memory and initialize
	int threadsPerBlock = 256;
	int blocksPerGrid = (4*Nxs*Nys+threadsPerBlock-1)/threadsPerBlock;
	// dim3 threadsPerBlock(threadsPerBlock_,threadsPerBlock_,1);
	// dim3 blocksPerGrid(blocksPerGrid_,blocksPerGrid_,1);
	initialize<<<blocksPerGrid,threadsPerBlock>>>(4*Nxs*Nys,sol);
	initialize<<<blocksPerGrid,threadsPerBlock>>>(4*(Nxs-2)*(Nys-2),sol_old);
	initialize<<<blocksPerGrid,threadsPerBlock>>>(4*(Nxs-2)*(Nys-2),res);
	hipDeviceSynchronize();
	
  // Insert physical boundary conditions
	physical_boundary_conditions<<<blocksPerGrid,threadsPerBlock>>>(Nxs,Nys,sol,T_left,T_right,T_up,T_down);
	hipDeviceSynchronize();
	
  // Obtain solution
  parallel_solver(Nxs,Nys,sol,sol_old,res,iter,beta,eps,blocksPerGrid,threadsPerBlock);
	hipDeviceSynchronize();

  // Print results to file
  FILE *results;
  results = fopen("results.txt","w");
  fprintf(results,"%.8f %.8f ",Lx,Ly);
  for(int i=2;i<Nx;i++)
    fprintf(results,"%.8f ",0.);
  fprintf(results,"\n");
	
	for(int j=Nys-1;j>0;j--){
		for(int i=0;i<Nxs-1;i++)
			fprintf(results,"%.8f ",sol[0*Nxs*Nys+Nys*j+i]);
		for(int i=1;i<Nxs;i++)
			fprintf(results,"%.8f ",sol[1*Nxs*Nys+Nys*j+i]);
		
		fprintf(results,"\n");
	}
	for(int j=Nys-2;j>=0;j--){
		for(int i=0;i<Nxs-1;i++)
			fprintf(results,"%.8f ",sol[2*Nxs*Nys+Nys*j+i]);
		for(int i=1;i<Nxs;i++)
			fprintf(results,"%.8f ",sol[3*Nxs*Nys+Nys*j+i]);
		
		fprintf(results,"\n");
	}
  fclose(results);
	
	// Free memories
	hipFree(sol);
	hipFree(sol_old);
	hipFree(res);
	// free(sol_host);
	// free(sol_old_host);
	// delete[] sol;
	// delete[] sol_old;
	
	time(&end);
	double time_taken = double(end - start);
	printf("\n%f s\n",time_taken);
	
	puts("end");

  return 0;
}