
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<ctime>
// #include<iostream>

/*
v2 difference: variable domain division
*/

void print_2d_array(int Nx,int Ny,int k,double *M);
__global__
void initialize(int size,double *M);
__device__
void solver(int Nx,int Ny,double *sol,int k,double beta);
void physical_boundary_conditions(int Nxs,int Nys,int divX,int divY,double *sol,
																	double T_left,double T_right,double T_up,
																	double T_down);
void vertical_phantom_BC(int Nxs,int Nys,double *sol,int k);
void horizontal_phantom_BC(int Nxs,int Nys,int divX,double *sol,int k);
int element_in_array(int size,int *array,int num);
void phantom_boundary_conditions(int Nxs,int Nys,int divX,int divY,double *sol,
																 int *jump_points);
__device__
void save_sol_old(int Nxs,int Nys,double *sol,double *sol_old,int size);
__device__
void get_residuals(int Nxs,int Nys,int divX,int divY,double *sol,double *sol_old
									 ,double *res);
double sum_residuals(int Nxs,int Nys,double *res);
__global__ 
void iterate_once(int Nxs,int Nys,int divX,int divY,double *sol,double *sol_old,
									double *res,double beta,int size);
void get_jump_points(int divX,int divY,int *jump_points);
void parallel_solver(int Nxs,int Nys,int divX,int divY,double *sol,
										 double *sol_old,double *res,int iter,double beta,
										 double eps,int BpG,int TpB);
void print_results(int Nx,int Ny,int Nxs,int Nys,int divX,int divY,double Lx,
									 double Ly,double *sol);

int main(){
	time_t start,end;
	time(&start);
	
	// Problem parameters
  double Lx = 1., Ly = 1.;
  double T_down = -0., T_up = -100.;
  double T_left = 50., T_right = 50.;

  // Mesh parameters
  int Nx = 402, Ny = 402; // Complete domain (even values)
  double deltaX = Lx/((double) Nx);
  double deltaY = Ly/((double) Ny);
  double beta = deltaX/deltaY;
	
	// Numeric solution configuration
  int iter = 1000000;
  double eps = 1e-3;
	int divX = 10, divY = 10; // Number of domain divisions
	if((Nx-2)%divX != 0){
		puts("Error: (Nx-2) must be divisible by divX");
		return 1;
	}
	if((Ny-2)%divY != 0){
		puts("Error: (Ny-2) must be divisible by divY");
		return 2;
	}
	
	// Allocate relevant arrays in GPU memory
	Nx = (int) (Nx/2)*2; // Ensure values are even
	Ny = (int) (Ny/2)*2;
	int Nxs = (Nx-2)/divX+2; // Dimensions for the submatrices
	int Nys = (Ny-2)/divY+2;
	if(Nxs<3 || Nys<3){
		puts("Error: Nxs and Nys must be equal or larger than 3. Redefine mesh \
				 configurations");
		return 3;
	}
	double *sol = new double[Nxs*Nys*divX*divY];
	double *sol_old = new double[(Nxs-2)*(Nys-2)*divX*divY];
	double *res = new double[(Nxs-2)*(Nys-2)*divX*divY];
	hipMallocManaged(&sol,sizeof(double)*Nxs*Nys*divX*divY);
	hipMallocManaged(&sol_old,sizeof(double)*(Nxs-2)*(Nys-2)*divX*divY);
	hipMallocManaged(&res,sizeof(double)*(Nxs-2)*(Nys-2)*divX*divY);
	
	// Initialize arrays
	int threadsPerBlock = 1024;
	int blocksPerGrid = (Nxs*Nys*divX*divY+threadsPerBlock-1)/threadsPerBlock;
	initialize<<<blocksPerGrid,threadsPerBlock>>>(Nxs*Nys*divX*divY,sol);
	initialize<<<blocksPerGrid,threadsPerBlock>>>((Nxs-2)*(Nys-2)*divX*divY,
						sol_old);
	initialize<<<blocksPerGrid,threadsPerBlock>>>((Nxs-2)*(Nys-2)*divX*divY,res);
	hipDeviceSynchronize();
	
  // Insert physical boundary conditions
	physical_boundary_conditions(Nxs,Nys,divX,divY,sol,T_left,T_right,T_up,
															 T_down);
	
  // Obtain solution
  parallel_solver(Nxs,Nys,divX,divY,sol,sol_old,res,iter,beta,eps,blocksPerGrid,
									threadsPerBlock);

  // Print results to file
  print_results(Nx,Ny,Nxs,Nys,divX,divY,Lx,Ly,sol);
	
	// Free memories
	hipFree(sol);
	hipFree(sol_old);
	hipFree(res);
	
	time(&end);
	double time_taken = double(end - start);
	printf("\n%f s\n",time_taken);

	puts("end");

  return 0;
}

void print_2d_array(int Nx,int Ny,int k,double *M){
  for(int j=Ny-1;j>=0;j--){
    for(int i=0;i<Nx;i++)
      printf("%f ",M[k*Nx*Ny+Ny*j+i]);

    putchar('\n');
  }
}

// __device__
// void dev_print_2d_array(int Nx,int Ny,int k,double *M){
  // for(int j=Ny-1;j>=0;j--){
    // for(int i=0;i<Nx;i++)
      // printf("%f ",M[k*Nx*Ny+Ny*j+i]);

    // printf("\n");
  // }
// }

// void print_2d_array(int Nx,int Ny,int k,double *M){
  // for(int j=0;j<Ny;j++){
    // for(int i=0;i<Nx;i++)
      // printf("%f ",M[k*Nx*Ny+Ny*j+i]);

    // putchar('\n');
  // }
// }

__global__
void initialize(int size,double *M){
	int index = blockIdx.x*blockDim.x + threadIdx.x;
  int stride = blockDim.x*gridDim.x;
  for(int i=index;i<size;i+=stride){
    M[i] = 0;
	}
}

__device__
void solver(int Nx,int Ny,double *sol,int k,double beta){
	if(blockIdx.x*blockDim.x + threadIdx.x == k){
		for(int j=1;j<Ny-1;j++){
			for(int i=1;i<Nx-1;i++)
				sol[k*Nx*Ny+Ny*j+i] = 1./(2.*(1.+beta*beta))*(sol[k*Nx*Ny+Ny*j+(i+1)] + 
															sol[k*Nx*Ny+Ny*j+(i-1)] + 
															beta*beta*sol[k*Nx*Ny+Ny*(j+1)+i] + 
															beta*beta*sol[k*Nx*Ny+Ny*(j-1)+i]);
		}
	}
}

// Regions are considered as below:
// | [0] [1] |
// | [2] [3] | 
void physical_boundary_conditions(int Nxs,int Nys,int divX,int divY,double *sol,
																	double T_left,double T_right,double T_up,
																	double T_down){
	
	// Left and right
	for(int k1=0,k2=divX-1;k1<=divY*(divX-1);k1+=divX,k2+=divX){
		for(int j=0;j<Nys;j++){
			sol[k1*Nxs*Nys+Nys*j+0] = T_left;
			sol[k2*Nxs*Nys+Nys*j+(Nxs-1)] = T_right;
		}
	}
	
	// Up and down
	for(int k1=0,k2=divX*divY-1;k1<divX;k1++,k2--){
		for(int i=0;i<Nxs;i++){
			sol[k1*Nxs*Nys+Nys*(Nys-1)+i] = T_up;
			sol[k2*Nxs*Nys+Nys*0+i] = T_down;
		}
	}
}

void vertical_phantom_BC(int Nxs,int Nys,double *sol,int k){
	for(int j=1;j<Nys-1;j++){
		// Left to right <-
		sol[(k+1)*Nxs*Nys+Nys*j+0] = sol[k*Nxs*Nys+Nys*j+(Nxs-2)];			
		
		// Right to left
		sol[k*Nxs*Nys+Nys*j+(Nxs-1)] = sol[(k+1)*Nxs*Nys+Nys*j+1];

	}
}

void horizontal_phantom_BC(int Nxs,int Nys,int divX,double *sol,int k){
	for(int i=1;i<Nxs-1;i++){
		// Lower to upper
		sol[k*Nxs*Nys+Nys*0+i] = sol[(k+divX)*Nxs*Nys+Nys*(Nys-2)+i];
		
		// Upper to lower
		sol[(k+divX)*Nxs*Nys+Nys*(Nys-1)+i] = sol[k*Nxs*Nys+Nys*1+i];
	}
}

int element_in_array(int size,int *array,int num){
	for(int i=0;i<size;i++){
		if(array[i] == num)
			return 1;
	}
	return 0;
}

void phantom_boundary_conditions(int Nxs,int Nys,int divX,int divY,double *sol,
																 int *jump_points){
	// Vertical
	for(int k=0;k<divY*divX-1;k++){
		if(element_in_array(divY-1,jump_points,k))
			k++;
		// printf("%d\n",k);
		vertical_phantom_BC(Nxs,Nys,sol,k);
	}
			
	// Horizontal
	for(int k=0;k<divX*(divY-1);k++){
		horizontal_phantom_BC(Nxs,Nys,divX,sol,k);
	}
}

__device__
void save_sol_old(int Nxs,int Nys,double *sol,double *sol_old,int size){
	int index = blockIdx.x*blockDim.x + threadIdx.x;
  int stride = blockDim.x*gridDim.x;
	int k;
	int row;
	int col;
	
	for(int i=index;i<(Nxs-2)*(Nys-2)*size;i+=stride){
		k = i/((Nxs-2)*(Nys-2));
		row = (i - k*(Nxs-2)*(Nys-2))/(Nxs-2);
		col = i - k*(Nxs-2)*(Nys-2) - (Nys-2)*row;
		sol_old[i] = sol[k*Nxs*Nys+Nys*(row+1)+(col+1)];
	}
}

__device__
void get_residuals(int Nxs,int Nys,int divX,int divY,double *sol,double *sol_old
									 ,double *res){
	int index = blockIdx.x*blockDim.x + threadIdx.x;
  int stride = blockDim.x*gridDim.x;
	int k;
	int row;
	int col;
	for(int i=index;i<(Nxs-2)*(Nys-2)*divX*divY;i+=stride){
		k = i/((Nxs-2)*(Nys-2));
		row = (i - k*(Nxs-2)*(Nys-2))/(Nxs-2);
		col = i - k*(Nxs-2)*(Nys-2) - (Nys-2)*row;
		res[i] = fabs(sol[k*Nxs*Nys+Nys*(row+1)+(col+1)] - 
							 sol_old[i]);
	}
}

double sum_residuals(int Nxs,int Nys,int divX,int divY,double *res){
	double res_sum = 0.;
	for(int i=0;i<Nxs*Nys*divX*divY;i++)
		res_sum += res[i];
	
	return res_sum;
}

__global__ 
void iterate_once(int Nxs,int Nys,int divX,int divY,double *sol,double *sol_old,
									double *res,double beta,int size){
	int index = blockIdx.x*blockDim.x + threadIdx.x;
  int stride = blockDim.x*gridDim.x;
	save_sol_old(Nxs,Nys,sol,sol_old,size); // Save values - convergence checking
	for(int k=index;k<size;k+=stride)
		solver(Nxs,Nys,sol,k,beta);
	
	get_residuals(Nxs,Nys,divX,divY,sol,sol_old,res);
}

void get_jump_points(int divX,int divY,int *jump_points){
	for(int i=0;i<divY-1;i++)
		jump_points[i] = (i+1)*divX-1;
}

void parallel_solver(int Nxs,int Nys,int divX,int divY,double *sol,
										 double *sol_old,double *res,int iter,double beta,
										 double eps,int BpG,int TpB){
	double res_val;
	int *jump_points = new int[divY-1];
	hipMallocManaged(&jump_points,sizeof(int)*(divY-1));
	get_jump_points(divX,divY,jump_points);
	hipDeviceSynchronize();
	
	for(int loop=0;loop<iter;loop++){
		// Calculate one iteration for each region
		iterate_once<<<BpG,TpB>>>(Nxs,Nys,divX,divY,sol,sol_old,res,beta,divX*divY);
		hipDeviceSynchronize();
		
		// Check convergence
		res_val = sum_residuals(Nxs-2,Nys-2,divX,divY,res);
		if(loop > 0){
			if(res_val <= eps){
				puts("Convergence!");
				break;
			}
		}
		
		printf("Iteration %d | Residuals = %f\n",loop+1,res_val);
		
		// Update phantom boundary conditions
		phantom_boundary_conditions(Nxs,Nys,divX,divY,sol,jump_points);
	}
	
	hipFree(jump_points);
}

void print_results(int Nx,int Ny,int Nxs,int Nys,int divX,int divY,double Lx,
									 double Ly,double *sol){
	FILE *results;
  results = fopen("results.txt","w");
  fprintf(results,"%.8f %.8f ",Lx,Ly);
  for(int i=2;i<Nx;i++)
    fprintf(results,"%.8f ",0.);
  fprintf(results,"\n");
	
	int istart,iend;
	int jstart,jend;
	for(int row=0;row<divY;row++){
		
		if(row==0){
			jstart = Nys-1;
			jend = 1;
		}
		else if(row==divY-1){
			jstart = Nys-2;
			jend = 0;
		}
		else{
			jstart = Nys-2;
			jend = 1;
		}
			
		for(int j=jstart;j>=jend;j--){
			for(int col=0;col<divX;col++){
				if(col==0){
					istart = 0;
					iend = Nxs-2;
				}
				else if(col==divX-1){
					istart = 1;
					iend = Nxs-1;
				}
				else{
					istart = 1;
					iend = Nxs-2;
				}
				for(int i=istart;i<=iend;i++)
					fprintf(results,"%.8f ",sol[(row*divY+col)*Nxs*Nys+Nys*j+i]);
				
				// fprintf(results," | ");
			}
			fprintf(results,"\n");
		}
		// fprintf(results,"-------------------------------\n");
	}
	fclose(results);
}